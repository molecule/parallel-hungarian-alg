#include "hip/hip_runtime.h"
#include "HungarianAlg.h"
#define NUM_THREADS 256

using namespace std;

int parallel = 1;
int DEBUG = 0;
__device__ double d_columnAnswer;
__device__ double d_rowAnswer;

AssignmentProblemSolver::AssignmentProblemSolver()
{
}

AssignmentProblemSolver::~AssignmentProblemSolver()
{
}

//
//  timer
//
double read_timer( )
{
    static bool initialized = false;
    static struct timeval start;
    struct timeval end;
    if( !initialized )
    {   
        gettimeofday( &start, NULL );
        initialized = true;
    }   
    gettimeofday( &end, NULL );
    return (end.tv_sec - start.tv_sec) + 1.0e-6 * (end.tv_usec - start.tv_usec);
}

//
//  command line option processing
//
int find_option( int argc, char **argv, const char *option )
{
    for( int i = 1; i < argc; i++ )
        if( strcmp( argv[i], option ) == 0 ) 
            return i;
    return -1; 
}

int read_int( int argc, char **argv, const char *option, int default_value )
{
    int iplace = find_option( argc, argv, option );
    if( iplace >= 0 && iplace < argc-1 )
        return atoi( argv[iplace+1] );
    return default_value;
}

double AssignmentProblemSolver::Solve(vector<vector<double> >& DistMatrix,vector<int>& Assignment,TMethod Method)
{
    if(DEBUG) {
        printf("solve\n");
    }
    int N=DistMatrix.size(); // number of columns (tracks)
    int M=DistMatrix[0].size(); // number of rows (measurements)

    int *assignment		=new int[N];
    double *distIn		=new double[N*M];

    double  cost;
    // Fill matrix with random numbers
    for(int i=0; i<N; i++)
    {
        for(int j=0; j<M; j++)
        {
            distIn[i+N*j] = DistMatrix[i][j];
        }
    }
    switch(Method)
    {
        case optimal: assignmentoptimal(assignment, &cost, distIn, N, M); break;

        case many_forbidden_assignments: assignmentoptimal(assignment, &cost, distIn, N, M); break;

        case without_forbidden_assignments: assignmentoptimal(assignment, &cost, distIn, N, M); break;
    }

    // form result
    Assignment.clear();
    for(int x=0; x<N; x++)
    {
        Assignment.push_back(assignment[x]);
    }

    delete[] assignment;
    delete[] distIn;
    return cost;
}
// --------------------------------------------------------------------------
// Computes the optimal assignment (minimum overall costs) using Munkres algorithm.
// --------------------------------------------------------------------------


__global__ void findMinCol_gpu(double* d_distMatrix, double* d_dualVariablesColumn, int n) {
    int tid = threadIdx.x * blockDim.x;
    if (tid >= n) return;
    int endIndex = tid + blockDim.x;

    d_columnAnswer = d_distMatrix[tid];
    for(int i = tid; i < endIndex; i++) {
        if (d_distMatrix[i] < d_columnAnswer) { d_columnAnswer = d_distMatrix[i]; }	
    }
    //printf("threadIdx.x: %d, tid: %d, endIndex: %d, d_colAnswer: %f\n", threadIdx.x, tid, endIndex, d_columnAnswer);
    d_dualVariablesColumn[threadIdx.x] = d_columnAnswer;

}

__global__ void findMinRow_gpu(double* d_distMatrix, double* d_dualVariablesRow, int n) {
    int tid = threadIdx.x;
    if (tid >= n) return;
    int endIndex = n;

    d_rowAnswer = d_distMatrix[tid];
    for(int i = tid; i < endIndex; i += blockDim.x) {
        if (d_distMatrix[i] < d_rowAnswer) { d_rowAnswer = d_distMatrix[i]; }	
    }
    //printf("tid: %d, endIndex: %d, d_rowAnswer: %f\n", tid, endIndex, d_rowAnswer);
    d_dualVariablesRow[threadIdx.x] = d_rowAnswer;
} 

__global__ void subtractMinElementRow_gpu(double* d_distMatrix, double* d_dualVariablesRow, int n) {
    int tid = threadIdx.x;
    if (tid >= n) return;
    //int endIndex = n; 

    // Subtract the smallest element in this row from each element in this row.
    int nOfRows = sqrt((float)n);
    int rowIdx = threadIdx.x % nOfRows;
    double before = d_distMatrix[tid];
    d_distMatrix[tid] = d_distMatrix[tid] - d_dualVariablesRow[rowIdx];
    printf("subtractMinElemRow, tid: %d, minElem: %f, bef: %f, aft: %f, rowIdx: %d \n", tid, d_dualVariablesRow[rowIdx], before, d_distMatrix[tid], rowIdx);
}

void AssignmentProblemSolver::assignmentoptimal(int *assignment, double *cost, double *distMatrixIn, int nOfRows, int nOfColumns)
{
    if (DEBUG) {
        printf("assignment optimal.\n");
    }
    double *distMatrix;
    double *dualVariablesRow;
    double *dualVariablesColumn;
    double *distMatrixTemp;
    double *distMatrixEnd;
    double *columnEnd;
    double  value;
    double  minValue;

    bool *coveredColumns;
    bool *coveredRows;
    bool *starMatrix;
    bool *newStarMatrix;
    bool *primeMatrix;

    int nOfElements;
    int minDim;
    int row;
    int col;

    // Init
    *cost = 0;
    for(row=0; row<nOfRows; row++)
    {
        assignment[row] = -1.0;
    }

    // Generate distance matrix
    // and check matrix elements positiveness :)

    // Total elements number
    nOfElements   = nOfRows * nOfColumns;
    // Memory allocation
    distMatrix    = (double *)malloc(nOfElements * sizeof(double));
    double * d_distMatrix;
    hipMalloc((void **) &d_distMatrix, nOfElements * sizeof(double));

    dualVariablesRow = (double *)malloc(nOfRows * sizeof(double));
    double * d_dualVariablesRow;
    hipMalloc((void **) &d_dualVariablesRow, nOfRows * sizeof(double));
    dualVariablesColumn = (double *)malloc(nOfColumns * sizeof(double));
    double * d_dualVariablesColumn;
    hipMalloc((void**) &d_dualVariablesColumn, nOfColumns * sizeof(double));

    // Pointer to last element
    distMatrixEnd = distMatrix + nOfElements;

    //
    for(row=0; row<nOfElements; row++)
    {
        value = distMatrixIn[row];
        if(value < 0)
        {
            cout << "All matrix elements have to be non-negative." << endl;
        }
        distMatrix[row] = value;
        //printf("distMatrix[%d]: %f\n", row, value);
    }

    // Memory allocation
    coveredColumns = (bool *)calloc(nOfColumns,  sizeof(bool));
    coveredRows    = (bool *)calloc(nOfRows,     sizeof(bool));
    starMatrix     = (bool *)calloc(nOfElements, sizeof(bool));
    primeMatrix    = (bool *)calloc(nOfElements, sizeof(bool));
    newStarMatrix  = (bool *)calloc(nOfElements, sizeof(bool)); /* used in step4 */

    hipMemcpy(d_distMatrix, distMatrix, nOfElements * sizeof(double), hipMemcpyHostToDevice);
    //int blks = (nOfElements + NUM_THREADS - 1) / NUM_THREADS;
    //int blks = nOfRows;
    int blks = 1;
    //nOfRows = 1;
    //nOfColumns = 1;
    if (parallel) {
        //findMinCol_gpu <<< blks, nOfRows >>> (d_distMatrix, d_dualVariablesColumn, nOfElements);
        findMinRow_gpu <<< blks, nOfColumns >>> (d_distMatrix, d_dualVariablesRow, nOfElements);
        hipDeviceSynchronize(); // GPU doesn't block CPU thread
        subtractMinElementRow_gpu <<< blks, nOfElements >>> (d_distMatrix, d_dualVariablesRow, nOfElements);
        hipDeviceSynchronize(); // GPU doesn't block CPU thread

        //hipMemcpy(dualVariablesRow, d_dualVariablesRow, nOfRows * sizeof(double), hipMemcpyDeviceToHost);
        //hipMemcpy(dualVariablesColumn, d_dualVariablesColumn, nOfColumns * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(distMatrix, d_distMatrix, nOfElements * sizeof(double), hipMemcpyDeviceToHost);

        if (DEBUG) {
            for(int i = 0; i < nOfElements; i++) {
                printf("distMatrix[%d]: %f\n", i, distMatrix[i]);
            }
        //    for(int i = 0; i < nOfRows; i++) {
        //        printf("smallest value in row %d is: %f\n", i, dualVariablesRow[i]);
        //    }
            /*
            for(int i = 0; i < nOfColumns; i++) {
                printf("smallest value in column %d is: %f\n", i, dualVariablesColumn[i]);
            }
            */
        }
        return;
    } else {
    /* preliminary steps */
        minDim = nOfRows;
        for(row=0; row<nOfRows; row++)
        {
            /* find the smallest element in the row */
            distMatrixTemp = distMatrix + row;
            minValue = *distMatrixTemp;
            distMatrixTemp += nOfRows;
            while(distMatrixTemp < distMatrixEnd)
            {
                value = *distMatrixTemp;
                if(value < minValue)
                {
                    minValue = value;
                }
                distMatrixTemp += nOfRows;
            }
            /* subtract the smallest element from each element of the row */
            distMatrixTemp = distMatrix + row;
            while(distMatrixTemp < distMatrixEnd)
            {
                *distMatrixTemp -= minValue;
                distMatrixTemp += nOfRows;
            }
        }
        /* Steps 1 and 2a */
        for(row=0; row<nOfRows; row++)
        {
            for(col=0; col<nOfColumns; col++)
            {
                if(distMatrix[row + nOfRows*col] == 0)
                {
                    if(!coveredColumns[col])
                    {
                        starMatrix[row + nOfRows*col] = true;
                        coveredColumns[col]           = true;
                        break;
                    }
                }
            }
        }
    }
    /* move to step 2b */
    step2b(assignment, distMatrix, starMatrix, newStarMatrix, primeMatrix, coveredColumns, coveredRows, nOfRows, nOfColumns, minDim);
    /* compute cost and remove invalid assignments */
    computeassignmentcost(assignment, cost, distMatrixIn, nOfRows);
    /* free allocated memory */
    free(distMatrix);
    free(coveredColumns);
    free(coveredRows);
    free(starMatrix);
    free(primeMatrix);
    free(newStarMatrix);
    return;
}
// --------------------------------------------------------------------------
//
// --------------------------------------------------------------------------
void AssignmentProblemSolver::buildassignmentvector(int *assignment, bool *starMatrix, int nOfRows, int nOfColumns)
{
    if (DEBUG) {printf("build assignment vector.\n");}
    int row, col;
    for(row=0; row<nOfRows; row++)
    {
        for(col=0; col<nOfColumns; col++)
        {
            if(starMatrix[row + nOfRows*col])
            {
                assignment[row] = col;
                break;
            }
        }
    }
}
// --------------------------------------------------------------------------
//
// --------------------------------------------------------------------------
void AssignmentProblemSolver::computeassignmentcost(int *assignment, double *cost, double *distMatrix, int nOfRows)
{
    if (DEBUG) { printf("compute assignment cost.\n");}
    int row, col;
    for(row=0; row<nOfRows; row++)
    {
        col = assignment[row];
        if(col >= 0)
        {
            *cost += distMatrix[row + nOfRows*col];
        }
    }
}

// --------------------------------------------------------------------------
//
// --------------------------------------------------------------------------
void AssignmentProblemSolver::step2a(int *assignment, double *distMatrix, bool *starMatrix, bool *newStarMatrix, bool *primeMatrix, bool *coveredColumns, bool *coveredRows, int nOfRows, int nOfColumns, int minDim)
{
    if (DEBUG) { printf("step 2a\n"); }
    bool *starMatrixTemp, *columnEnd;
    int col;
    /* cover every column containing a starred zero */
    for(col=0; col<nOfColumns; col++)
    {
        starMatrixTemp = starMatrix     + nOfRows*col;
        columnEnd      = starMatrixTemp + nOfRows;
        while(starMatrixTemp < columnEnd)
        {
            if(*starMatrixTemp++)
            {
                coveredColumns[col] = true;
                break;
            }
        }
    }
    /* move to step 3 */
    step2b(assignment, distMatrix, starMatrix, newStarMatrix, primeMatrix, coveredColumns, coveredRows, nOfRows, nOfColumns, minDim);
}

// --------------------------------------------------------------------------
//
// --------------------------------------------------------------------------
void AssignmentProblemSolver::step2b(int *assignment, double *distMatrix, bool *starMatrix, bool *newStarMatrix, bool *primeMatrix, bool *coveredColumns, bool *coveredRows, int nOfRows, int nOfColumns, int minDim)
{
    if (DEBUG) { printf("step 2b\n");}
    int col, nOfCoveredColumns;
    /* count covered columns */
    nOfCoveredColumns = 0;
    for(col=0; col<nOfColumns; col++)
    {
        if(coveredColumns[col])
        {
            nOfCoveredColumns++;
        }
    }
    if(nOfCoveredColumns == minDim)
    {
        /* algorithm finished */
        buildassignmentvector(assignment, starMatrix, nOfRows, nOfColumns);
    }
    else
    {
        /* move to step 3 */
        step3(assignment, distMatrix, starMatrix, newStarMatrix, primeMatrix, coveredColumns, coveredRows, nOfRows, nOfColumns, minDim);
    }
}

// --------------------------------------------------------------------------
//
// --------------------------------------------------------------------------
void AssignmentProblemSolver::step3(int *assignment, double *distMatrix, bool *starMatrix, bool *newStarMatrix, bool *primeMatrix, bool *coveredColumns, bool *coveredRows, int nOfRows, int nOfColumns, int minDim)
{
    if (DEBUG) {printf("step 3\n");}
    bool zerosFound;
    int row, col, starCol;
    zerosFound = true;
    while(zerosFound)
    {
        zerosFound = false;
        for(col=0; col<nOfColumns; col++)
        {
            if(!coveredColumns[col])
            {
                for(row=0; row<nOfRows; row++)
                {
                    if((!coveredRows[row]) && (distMatrix[row + nOfRows*col] == 0))
                    {
                        /* prime zero */
                        primeMatrix[row + nOfRows*col] = true;
                        /* find starred zero in current row */
                        for(starCol=0; starCol<nOfColumns; starCol++)
                            if(starMatrix[row + nOfRows*starCol])
                            {
                                break;
                            }
                        if(starCol == nOfColumns) /* no starred zero found */
                        {
                            /* move to step 4 */
                            step4(assignment, distMatrix, starMatrix, newStarMatrix, primeMatrix, coveredColumns, coveredRows, nOfRows, nOfColumns, minDim, row, col);
                            return;
                        }
                        else
                        {
                            coveredRows[row]        = true;
                            coveredColumns[starCol] = false;
                            zerosFound              = true;
                            break;
                        }
                    }
                }
            }
        }
    }
    /* move to step 5 */
    step5(assignment, distMatrix, starMatrix, newStarMatrix, primeMatrix, coveredColumns, coveredRows, nOfRows, nOfColumns, minDim);
}

// --------------------------------------------------------------------------
//
// --------------------------------------------------------------------------
void AssignmentProblemSolver::step4(int *assignment, double *distMatrix, bool *starMatrix, bool *newStarMatrix, bool *primeMatrix, bool *coveredColumns, bool *coveredRows, int nOfRows, int nOfColumns, int minDim, int row, int col)
{
    if (DEBUG) { printf("step 4\n");}
    int n, starRow, starCol, primeRow, primeCol;
    int nOfElements = nOfRows*nOfColumns;
    /* generate temporary copy of starMatrix */
    for(n=0; n<nOfElements; n++)
    {
        newStarMatrix[n] = starMatrix[n];
    }
    /* star current zero */
    newStarMatrix[row + nOfRows*col] = true;
    /* find starred zero in current column */
    starCol = col;
    for(starRow=0; starRow<nOfRows; starRow++)
    {
        if(starMatrix[starRow + nOfRows*starCol])
        {
            break;
        }
    }
    while(starRow<nOfRows)
    {
        /* unstar the starred zero */
        newStarMatrix[starRow + nOfRows*starCol] = false;
        /* find primed zero in current row */
        primeRow = starRow;
        for(primeCol=0; primeCol<nOfColumns; primeCol++)
        {
            if(primeMatrix[primeRow + nOfRows*primeCol])
            {
                break;
            }
        }
        /* star the primed zero */
        newStarMatrix[primeRow + nOfRows*primeCol] = true;
        /* find starred zero in current column */
        starCol = primeCol;
        for(starRow=0; starRow<nOfRows; starRow++)
        {
            if(starMatrix[starRow + nOfRows*starCol])
            {
                break;
            }
        }
    }
    /* use temporary copy as new starMatrix */
    /* delete all primes, uncover all rows */
    for(n=0; n<nOfElements; n++)
    {
        primeMatrix[n] = false;
        starMatrix[n]  = newStarMatrix[n];
    }
    for(n=0; n<nOfRows; n++)
    {
        coveredRows[n] = false;
    }
    /* move to step 2a */
    step2a(assignment, distMatrix, starMatrix, newStarMatrix, primeMatrix, coveredColumns, coveredRows, nOfRows, nOfColumns, minDim);
}

// --------------------------------------------------------------------------
//
// --------------------------------------------------------------------------
void AssignmentProblemSolver::step5(int *assignment, double *distMatrix, bool *starMatrix, bool *newStarMatrix, bool *primeMatrix, bool *coveredColumns, bool *coveredRows, int nOfRows, int nOfColumns, int minDim)
{
    if (DEBUG) { printf("step 5\n");}
    double h, value;
    int row, col;
    /* find smallest uncovered element h */
    h = DBL_MAX;
    for(row=0; row<nOfRows; row++)
    {
        if(!coveredRows[row])
        {
            for(col=0; col<nOfColumns; col++)
            {
                if(!coveredColumns[col])
                {
                    value = distMatrix[row + nOfRows*col];
                    if(value < h)
                    {
                        h = value;
                    }
                }
            }
        }
    }
    /* add h to each covered row */
    for(row=0; row<nOfRows; row++)
    {
        if(coveredRows[row])
        {
            for(col=0; col<nOfColumns; col++)
            {
                distMatrix[row + nOfRows*col] += h;
            }
        }
    }
    /* subtract h from each uncovered column */
    for(col=0; col<nOfColumns; col++)
    {
        if(!coveredColumns[col])
        {
            for(row=0; row<nOfRows; row++)
            {
                distMatrix[row + nOfRows*col] -= h;
            }
        }
    }
    /* move to step 3 */
    step3(assignment, distMatrix, starMatrix, newStarMatrix, primeMatrix, coveredColumns, coveredRows, nOfRows, nOfColumns, minDim);
}


// --------------------------------------------------------------------------
// Computes a suboptimal solution. Good for cases without forbidden assignments.
// --------------------------------------------------------------------------
void AssignmentProblemSolver::assignmentsuboptimal2(int *assignment, double *cost, double *distMatrixIn, int nOfRows, int nOfColumns)
{
    printf("assignmentsuboptimal2...............................................\n");
}

// --------------------------------------------------------------------------
// Computes a suboptimal solution. Good for cases with many forbidden assignments.
// --------------------------------------------------------------------------
void AssignmentProblemSolver::assignmentsuboptimal1(int *assignment, double *cost, double *distMatrixIn, int nOfRows, int nOfColumns)
{
    printf("assignmentsuboptimal1..............................................\n");
}
// --------------------------------------------------------------------------
// Usage example
// --------------------------------------------------------------------------
//int main(void)
int main( int argc, char **argv )
{
    int n = read_int( argc, argv, "-n", 10 );
    int m = read_int( argc, argv, "-m", 10);
    int print = read_int( argc, argv, "-p", 0);
    parallel = read_int( argc, argv, "-L", 1);
    DEBUG = read_int( argc, argv, "-d", 0);
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    // Matrix size
    int N=n; // tracks rows
    int M=m; // detects columns
    // Random numbers generator initialization
    srand (time(NULL));
    // Distance matrix N-th track to M-th detect.
    vector< vector<double> > Cost(N,vector<double>(M));
    // Fill matrix with random values
    printf("HungarianAlg.cpp\n");
    printf("Creating a random Cost Matrix:\n");
    for(int i=0; i<N; i++)
    {
        for(int j=0; j<M; j++)
        {
            Cost[i][j] = (double)(rand()%1000)/1000.0;
            if (print) { std::cout << Cost[i][j] << "\t";}
        }
        if (print) { std::cout << std::endl;}
    }
    AssignmentProblemSolver APS;
    vector<int> Assignment;
    printf("Solving the random matrix...\n");
    double solve_time = read_timer( );
    double totalCost = APS.Solve(Cost, Assignment);
    //cout << APS.Solve(Cost,Assignment) << endl;
    solve_time = read_timer( ) - solve_time;
    printf("Total solve_time: %g\n", solve_time);

    // Output the result
    if (print) {
        for(int x=0; x<N; x++)
        {
            std::cout << x << ":" << Assignment[x] << "\t";
        }
    }
}
// --------------------------------------------------------------------------
